#include "hip/hip_runtime.h"
#include "common/types.h"
#include "functional/functional.h"
#include "functional/tensor.h"
#include "tensors/gpu/cuda_helpers.h"

#include <thrust/tuple.h>

namespace marian {

namespace gpu {



template <typename T>
__global__ void gAlibi(
  functional::Tensor<T> out,
  functional::Array<functional::Tensor<T>, 4> inputs,
  int numHeads,
  int start,
  float maskFactor,
  bool addCausalMask,
  int window  // new parameter for the window limit
) {

  constexpr size_t N = functional::Shape::size();
  functional::Array<int, N> oDims;
  int length = out.shape().elements();

  const auto& mask   = inputs[0];
  const auto& slopes = inputs[1];
  const auto& biases = inputs[2];
  const auto& shift  = inputs[3];

  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      out.shape().dims(index, oDims);

      int beamIdx      = oDims[0];
      int batchHeadIdx = oDims[1];
      int queryIdx     = oDims[2];
      int keyIdx       = oDims[3];

      // [[maybe_unused]] because NVCC seems to have a bug telling me the variable is not referenced when it appears in an intializer; this surpresses the warning.
      [[maybe_unused]] int batchIdx = batchHeadIdx / numHeads;
      [[maybe_unused]] int headIdx  = batchHeadIdx % numHeads;

      int keyPos       = keyIdx;
      int queryPos     = queryIdx + start;

      float relPos   = (float)keyPos - (float)queryPos;

      if(shift.data() != nullptr)
        relPos -= (float)shift[{beamIdx, batchIdx, queryIdx, 0}];

      float slope = (float)slopes[{0, headIdx, 0, 0}];
      float bias  = (float)biases[{0, headIdx, 0, 0}];
      float signedAlibi = relPos + bias;
      float alibi = -abs(slope) * abs(signedAlibi); // range (-inf, 0)

      float binMask = (float)mask[{0, batchIdx, keyIdx, 0}];
      // Only apply the window check if window != 0.
      if(window > 0 && abs(relPos) > window)
        binMask = 0.f; // set binMask to 0 if outside the window

      float logMask = binMask == 0 ? -maskFactor : maskFactor; // range (-maskFactor, maskFactor)

      if(addCausalMask) {
        float causalMask = keyPos > queryPos ? -maskFactor : maskFactor; // range (-maskFactor, maskFactor)
        logMask          = min(logMask, causalMask); // range (-maskFactor, maskFactor) if any mask is set to -maskFactor then the result is -maskFactor
      }

      out[index] = (T)min(logMask, alibi);
    }
  }
}

template <class... Tensors>
void Alibi(int numHeads, int start, bool addCausalMask, Tensor out, Tensors... tensors) {
  hipSetDevice(out->getDeviceId().no);
  int length = out->size();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  float maskFactor = std::numeric_limits<float>::infinity();

  // Get the ALIBI_WINDOW environment variable (default is 512).
  int window = 0;
  char* envWindow = std::getenv("ALIBI_WINDOW");
  if(envWindow)
    window = std::atoi(envWindow);

  constexpr size_t K = sizeof...(tensors);
  if(out->type() == Type::float32) {
    functional::Array<functional::Tensor<float>, K> inputs = {tensors...};
    gAlibi<float><<<blocks, threads>>>(out, inputs, numHeads, start, maskFactor, addCausalMask, window);
#if COMPILE_FP16
  } else if(out->type() == Type::float16) {
    functional::Array<functional::Tensor<half>, K> inputs = {tensors...};
    gAlibi<half><<<blocks, threads>>>(out, inputs, numHeads, start, maskFactor, addCausalMask, window);
#endif
  } else {
    ABORT("Alibi for type {} not implemented", out->type());
  }
}

// template specialization for h/cpp separation
template void Alibi<marian::Tensor, marian::Tensor, marian::Tensor, marian::Tensor >(int, int, bool, marian::Tensor, marian::Tensor, marian::Tensor, marian::Tensor, marian::Tensor);

template <typename T>
__global__ void gAlibiGrad(
  functional::Tensor<T> slopesGrad,
  functional::Tensor<T> biasesGrad,
  functional::Array<functional::Tensor<T>, 5> inputs,
  int numHeads,
  int start,
  bool addCausalMask,
  int window            // new parameter for the window limit
) {
  const auto& mask   = inputs[0];
  const auto& slopes = inputs[1];
  const auto& biases = inputs[2];
  const auto& shift  = inputs[3];
  const auto& adj    = inputs[4];

  int cols = adj.size() / numHeads;

  functional::Shape fullShape = adj.shape();
  int dimBeam      = fullShape[0];
  int dimBatchHead = fullShape[1];
  [[maybe_unused]] int dimBatch = dimBatchHead / numHeads;
  int dimQuery     = fullShape[2];
  int dimKeys      = fullShape[3];

  using A5 = functional::Array<int, 5>;
  using S5 = functional::ConstantShape<5>;
  S5 fullShape5(A5({dimBeam, dimBatch, numHeads, dimQuery, dimKeys}));
  S5 headShape5(A5({dimBeam, dimBatch, 1, dimQuery, dimKeys}));

  A5 dims5;
  const int HEAD_DIM = 2;

  // compute single element derivative for slopes and biases
  auto dJ_dxy = [&](int headIdx, int colIdx) -> thrust::tuple<float, float> {
    // get the location for one head
    headShape5.dims(colIdx, dims5);
    dims5[HEAD_DIM] = headIdx;
    int index = fullShape5.index(dims5);
    float vadj = (float)adj[index];

    int beamIdx  = dims5[0];
    int batchIdx = dims5[1];
    int queryIdx = dims5[3];
    int keyIdx   = dims5[4];

    int keyPos   = keyIdx;
    int queryPos = queryIdx + start;

    float relPos = (float)keyPos - (float)queryPos;
    if(shift.data() != nullptr)
      relPos -= (float)shift[{beamIdx, batchIdx, queryIdx, 0}];

    float slope = (float)slopes[{0, headIdx, 0, 0}];
    float bias  = (float)biases[{0, headIdx, 0, 0}];
    float binMask = (float)mask[{0, batchIdx, keyIdx, 0}];

    float signedAlibi = relPos + bias;

    // Incorporate the window: if window is active and the absolute value exceeds window, then gradients are zero.
    if(window > 0 && abs(relPos) > window) {
      binMask = 0.f;
    }

    // If causal masking is enabled.
    if(addCausalMask) {
      float causalMask = keyPos > queryPos ? 0.f : 1.f;
      binMask *= causalMask;
    }

    // compute derivative of slope
    float dslope = binMask * - 1.f * abs(signedAlibi) * vadj;
    if(slope < 0)
      dslope = -dslope;

    // compute derivative of bias
    float dbias  = binMask * -abs(slope) * 1.f * vadj;
    if(signedAlibi < 0)
      dbias = -dbias;

    return { dslope, dbias };
  };

  for(int bid = 0; bid < numHeads; bid += gridDim.x) {
    int headIdx = bid + blockIdx.x;
    if(headIdx < numHeads) {
      // get and assign shared memory
      extern __shared__ uint8_t _sharedBytes[];
      float* _sum = (float*)(_sharedBytes);
      auto sharedSlopes = [_sum](int idx) -> float& { return _sum[2 * idx + 0]; }; // use even indices for slopes
      auto sharedBiases = [_sum](int idx) -> float& { return _sum[2 * idx + 1]; }; // use odd indices for biases

      sharedSlopes(threadIdx.x) = 0.0;
      sharedBiases(threadIdx.x) = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int colIdx = tid + threadIdx.x;
        if(colIdx < cols) {
          float dslopes = 0, dbiases = 0;
          // get the element-wise derivative
          thrust::tie(dslopes, dbiases) = dJ_dxy(headIdx, colIdx);
          // accumulate by thread id
          sharedSlopes(threadIdx.x) += dslopes;
          sharedBiases(threadIdx.x) += dbiases;
        }
      }
      __syncthreads();

      // accumulate here over matrix dimensions, tree reduction
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          sharedSlopes(threadIdx.x) += sharedSlopes(threadIdx.x + skip); // float
          sharedBiases(threadIdx.x) += sharedBiases(threadIdx.x + skip); // float
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();

      // assign accumulated gradients here (preserving existing gradients)
      slopesGrad[headIdx] += (T)sharedSlopes(0);
      biasesGrad[headIdx] += (T)sharedBiases(0);
    }
    __syncthreads();
  }
}

template <typename T, class... Tensors>
void TypedAlibiGrad(int numHeads, int start, bool addCausalMask,
                    Tensor slopesGrad, Tensor biasesGrad, Tensors... tensors) {
  hipSetDevice(slopesGrad->getDeviceId().no);

  constexpr size_t K = sizeof...(tensors);
  functional::Array<functional::Tensor<T>, K> inputs = {tensors...};

  const auto& adj = inputs[K - 1]; // last one is adjoint and full broadcast shape
  int total = adj.size();

  // we will reduce over each head
  int blocks  = std::min(MAX_BLOCKS,  numHeads);
  int threads = std::min(MAX_THREADS, total / numHeads);
  int shared  = sizeof(float) * threads * 2; // Use float32 as accumulation type, we accumulate slopes and biases

  // Get the ALIBI_WINDOW environment variable (default to 0 if not set)
  int window = 0;
  char* envWindow = std::getenv("ALIBI_WINDOW");
  if(envWindow)
    window = std::atoi(envWindow);

  gAlibiGrad<T><<<blocks, threads, shared>>>(slopesGrad, biasesGrad, inputs, numHeads, start, addCausalMask, window);
}

template <class... Tensors>
void AlibiGrad(int numHeads, int start, bool addCausalMask, Tensor slopesGrad, Tensor biasesGrad, Tensors... tensors) {
  if(slopesGrad->type() == Type::float32) {
    TypedAlibiGrad<float>(numHeads, start, addCausalMask, slopesGrad, biasesGrad, tensors...);
#if COMPILE_FP16
  } else if(slopesGrad->type() == Type::float16) {
    TypedAlibiGrad<half>(numHeads, start, addCausalMask, slopesGrad, biasesGrad, tensors...);
#endif
  } else {
    ABORT("AlibiGrad for type {} not implemented", slopesGrad->type());
  }
}

// template specialization for h/cpp separation
template void AlibiGrad<marian::Tensor, marian::Tensor, marian::Tensor, marian::Tensor, marian::Tensor>(int, int, bool, marian::Tensor, marian::Tensor, marian::Tensor, marian::Tensor, marian::Tensor, marian::Tensor, marian::Tensor);
}
}
